#include "hip/hip_runtime.h"
#include <iostream>
#include <sstream>
#include <fstream>
#include <string>
#include <thrust/device_vector.h>
#include <thrust/sort.h>


using namespace std;

/**********************************************************
* **********************************************************
* error checking stufff
***********************************************************
***********************************************************/
// Enable this for error checking

#define CUDA_CHECK_ERROR
#define CudaSafeCall( err ) __cudaSafeCall( err, __FILE__, __LINE__ )
#define CudaCheckError() __cudaCheckError( __FILE__, __LINE__ )

inline void __cudaSafeCall( hipError_t err, const char *file, const int line )
{
    #ifdef CUDA_CHECK_ERROR
    #pragma warning( push )
    #pragma warning( disable: 4127 ) // Prevent warning on do-while(0);
    do
    {
        if ( hipSuccess != err )
        {
            fprintf( stderr,
            "cudaSafeCall() failed at %s:%i : %s\n",
            file, line, hipGetErrorString( err ) );
            exit( -1 );
        }
    } while ( 0 );

    #pragma warning( pop )
    #endif // CUDA_CHECK_ERROR
    return;
}
inline void __cudaCheckError( const char *file, const int line )
{
    #ifdef CUDA_CHECK_ERROR
    #pragma warning( push )
    #pragma warning( disable: 4127 ) // Prevent warning on do-while(0);
    
    do
    {
        hipError_t err = hipGetLastError();
        if ( hipSuccess != err )
        {
            fprintf( stderr,
            "cudaCheckError() failed at %s:%i : %s.\n",
            file, line, hipGetErrorString( err ) );
            exit( -1 );
        }
        // More careful checking. However, this will affect performance.
        // Comment if not needed.
        err = hipDeviceSynchronize();
        if( hipSuccess != err )
        {
            fprintf( stderr,
            "cudaCheckError() with sync failed at %s:%i : %s.\n",
            file, line, hipGetErrorString( err ) );
            exit( -1 );
        }
    } while ( 0 );
    #pragma warning( pop )
    #endif // CUDA_CHECK_ERROR
    return;
}

/***************************************************************
* **************************************************************
* end of error checking stuff
****************************************************************
***************************************************************/
// function takes an array pointer, and the number of rows and cols in the array, and
// allocates and intializes the array to a bunch of random numbers
// Note that this function creates a 1D array that is a flattened 2D array
// to access data item data[i][j], you must can use data[(i*rows) + j]
thrust::device_vector<int> makeRandArray(const int size, const int seed)
{
    srand(seed);
    thrust::device_vector<int> array(size);

    for (int i = 0; i < size; i++)
    {
        array[i] = std::rand() % 1000000;
    }

    return array;
}


//*******************************//
// your kernel here!!!!!!!!!!!!!!!!!
//*******************************//

__global__ void matavgKernel( int *data, int size )
{
}


int main( int argc, char* argv[] )
{
    // int * array; // the poitner to the array of rands
    int size, seed; // values for the size of the array
    bool printSorted = false;
    // and the seed for generating
    // random numbers
    // check the command line args
    if( argc < 4 )
    {
        std::cerr << "usage: "
        << argv[0]
        << " [amount of random nums to generate] [seed value for rand]"
        << " [1 to print sorted array, 0 otherwise]"
        << std::endl;
        exit( -1 );
    }

    // convert cstrings to ints
    {
        std::stringstream ss1( argv[1] );
        ss1 >> size;
    }

    {
        std::stringstream ss1( argv[2] );
        ss1 >> seed;
    }

    {
        int sortPrint;
        std::stringstream ss1( argv[3] );
        ss1 >> sortPrint;
        if( sortPrint == 1 )
        printSorted = true;
    }
    // get the random numbers
    thrust::device_vector<int> array = makeRandArray(size, seed);
    /***********************************
    * create a cuda timer to time execution
    **********************************/
    hipEvent_t startTotal, stopTotal;
    float timeTotal;
    hipEventCreate(&startTotal);
    hipEventCreate(&stopTotal);
    hipEventRecord( startTotal, 0 );


    /***********************************
    * end of cuda timer creation
    **********************************/
   /////////////////////////////////////////////////////////////////////
    /////////////////////// YOUR CODE HERE ///////////////////////
    thrust::sort(array.begin(), array.end());
    CudaCheckError();
    /////////////////////////////////////////////////////////////////////
    /*
    * You need to implement your kernel as a function at the top of this file.
    * Here you must
    * 1) allocate device memory
    * 2) set up the grid and block sizes
    * 3) call your kenrnel
    * 4) get the result back from the GPU
    *
    *
    * to use the error checking code, wrap any cudamalloc functions as follows:
    * CudaSafeCall( hipMalloc( &pointer_to_a_device_pointer,
    * length_of_array * sizeof( int ) ) );
    * Also, place the following function call immediately after you call your kernel
    * ( or after any other cuda call that you think might be causing an error )
    * CudaCheckError();
    */
    /***********************************
    * Stop and destroy the cuda timer
    **********************************/
    hipEventRecord( stopTotal, 0 );
    hipEventSynchronize( stopTotal );
    hipEventElapsedTime( &timeTotal, startTotal, stopTotal );
    hipEventDestroy( startTotal );
    hipEventDestroy( stopTotal );
	
    /***********************************
    * end of cuda timer destruction
    **********************************/
    std::cerr << "Total time in seconds: "
    << timeTotal / 1000.0 << std::endl;

    if( printSorted ){
        ///////////////////////////////////////////////
        /// Your code to print the sorted array here //
        thrust::copy(array.begin(), array.end(), std::ostream_iterator<int>(std::cout, " "));
        std::cout << std::endl;
        ///////////////////////////////////////////////
    }
}