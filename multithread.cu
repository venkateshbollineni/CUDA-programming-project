#include "hip/hip_runtime.h"
#include <iostream>
#include <sstream>
#include <fstream>
#include <string>
#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include <iostream>
#include <hip/hip_runtime.h>
using namespace std;

/**********************************************************
* **********************************************************
* error checking stufff
***********************************************************
***********************************************************/
// Enable this for error checking

#define CUDA_CHECK_ERROR
#define CudaSafeCall( err ) __cudaSafeCall( err, __FILE__, __LINE__ )
#define CudaCheckError() __cudaCheckError( __FILE__, __LINE__ )

inline void __cudaSafeCall( hipError_t err, const char *file, const int line )
{
    #ifdef CUDA_CHECK_ERROR
    #pragma warning( push )
    #pragma warning( disable: 4127 ) // Prevent warning on do-while(0);
    do
    {
        if ( hipSuccess != err )
        {
            fprintf( stderr,
            "cudaSafeCall() failed at %s:%i : %s\n",
            file, line, hipGetErrorString( err ) );
            exit( -1 );
        }
    } while ( 0 );

    #pragma warning( pop )
    #endif // CUDA_CHECK_ERROR
    return;
}
inline void __cudaCheckError( const char *file, const int line )
{
    #ifdef CUDA_CHECK_ERROR
    #pragma warning( push )
    #pragma warning( disable: 4127 ) // Prevent warning on do-while(0);
    
    do
    {
        hipError_t err = hipGetLastError();
        if ( hipSuccess != err )
        {
            fprintf( stderr,
            "cudaCheckError() failed at %s:%i : %s.\n",
            file, line, hipGetErrorString( err ) );
            exit( -1 );
        }
        // More careful checking. However, this will affect performance.
        // Comment if not needed.
        err = hipDeviceSynchronize();
        if( hipSuccess != err )
        {
            fprintf( stderr,
            "cudaCheckError() with sync failed at %s:%i : %s.\n",
            file, line, hipGetErrorString( err ) );
            exit( -1 );
        }
    } while ( 0 );
    #pragma warning( pop )
    #endif // CUDA_CHECK_ERROR
    return;
}

/***************************************************************
* **************************************************************
* end of error checking stuff
****************************************************************
***************************************************************/

// function takes an array pointer, and the number of rows and cols in the array, and
// allocates and intializes the array to a bunch of random numbers
// Note that this function creates a 1D array that is a flattened 2D array
// to access data item data[i][j], you must can use data[(i*rows) + j]
int* makeRandArray(const int size, const int seed) {
    srand(seed);
    int *array = new int[size];
    for (int i = 0; i < size; ++i) {
        array[i] = std::rand() % 100000;
    }
    return array;
}


//*******************************//
// your kernel here!!!!!!!!!!!!!!!!!
//*******************************//
const int MAX_THREADS_PER_BLOCK = 1024;

__global__ void mergeSort(int* array, int* temp, int size) 
{
    
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    for (int width = 1; width < size; width *= 2) 
    {
        if (tid < size) 
        {
            int left = tid * 2 * width;
            int mid = min(left + width - 1, size - 1);
            int right = min(left + 2 * width - 1, size - 1);

            if (left <= right) 
            {
                int i = left;
                int j = mid + 1;
                int k = left;

                while (i <= mid && j <= right) 
                {
                    if (array[i] <= array[j]) 
                    {
                        temp[k++] = array[i++];
                    }
                     else 
                    {
                        temp[k++] = array[j++];
                    }
                }
                while (i <= mid) 
                {
                    temp[k++] = array[i++];
                }

                while (j <= right) 
                {
                    temp[k++] = array[j++];
                }

                for (i = left; i <= right; i++) 
                {
                    array[i] = temp[i];
                }
            }
        }
        __syncthreads();
    }
}


int main( int argc, char* argv[] )
{
    int * array; // the poitner to the array of rands
    int size, seed; // values for the size of the array
    bool printSorted = false;
    // and the seed for generating
    // random numbers
    // check the command line args
    if( argc < 4 )
    {
        std::cerr << "usage: "
        << argv[0]
        << " [amount of random nums to generate] [seed value for rand]"
        << " [1 to print sorted array, 0 otherwise]"
        << std::endl;
        exit( -1 );
    }

    // convert cstrings to ints
    {
        std::stringstream ss1( argv[1] );
        ss1 >> size;
    }

    {
        std::stringstream ss1( argv[2] );
        ss1 >> seed;
    }

    {
        int sortPrint;
        std::stringstream ss1( argv[3] );
        ss1 >> sortPrint;
        if( sortPrint == 1 )
        printSorted = true;
    }
    // get the random numbers
    array = makeRandArray(size, seed);

    int* d_array;
    int* d_temp;
    int numBlocks = (size + MAX_THREADS_PER_BLOCK - 1) / MAX_THREADS_PER_BLOCK;
    //memory allocation on device
    CudaSafeCall(hipMalloc((void**)&d_array, size * sizeof(int)));
    CudaSafeCall(hipMalloc((void**)&d_temp, size * sizeof(int)));

    //copying data from host to device
    CudaSafeCall((hipMemcpy(d_array, array, size * sizeof(int), hipMemcpyHostToDevice)));
    hipDeviceSynchronize();
    /***********************************
    * create a cuda timer to time execution
    **********************************/
    hipEvent_t startTotal, stopTotal;
    float timeTotal;
    hipEventCreate(&startTotal);
    hipEventCreate(&stopTotal);
    hipEventRecord( startTotal, 0 );


    /***********************************
    * end of cuda timer creation
    **********************************/
   /////////////////////////////////////////////////////////////////////
    /////////////////////// YOUR CODE HERE ///////////////////////
 
    
    mergeSort<<<numBlocks, MAX_THREADS_PER_BLOCK>>>(d_array, d_temp, size);
    CudaCheckError();
   



    /////////////////////////////////////////////////////////////////////
    /*
    * You need to implement your kernel as a function at the top of this file.
    * Here you must
    * 1) allocate device memory
    * 2) set up the grid and block sizes
    * 3) call your kenrnel
    * 4) get the result back from the GPU
    *
    *
    * to use the error checking code, wrap any cudamalloc functions as follows:
    * CudaSafeCall( hipMalloc( &pointer_to_a_device_pointer,
    * length_of_array * sizeof( int ) ) );
    * Also, place the following function call immediately after you call your kernel
    * ( or after any other cuda call that you think might be causing an error )
    * CudaCheckError();
    */
    /***********************************
    * Stop and destroy the cuda timer
    **********************************/
    hipEventRecord( stopTotal, 0 );
    hipEventSynchronize( stopTotal );
    hipEventElapsedTime( &timeTotal, startTotal, stopTotal );
    hipEventDestroy( startTotal );
    hipEventDestroy( stopTotal );
	
    /***********************************
    * end of cuda timer destruction
    **********************************/
    std::cerr << "Total time in seconds: "
    << timeTotal / 1000.0 << std::endl;

    hipMemcpy(array, d_array, size * sizeof(int), hipMemcpyDeviceToHost);
   
    if (printSorted)
    {
        for (int i = 0; i < size; ++i)
        {
            std::cout << array[i] << " ";
        }
        std::cout << std::endl;
    }
    // Cleaning up allocated memory
    delete[] array;
    hipFree(d_array);
    hipFree(d_temp);


    return 0;
}